
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <vector>

// Kernel
template<typename T>
__global__ void add_kernel(T *c, const T *a, const T *b, size_t n, size_t step) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x+step;
    for(size_t i =idx;i<n;i+=step){
        c[i] = a[i] + b[i];
    }
}

template<typename T>
void vector_add(T* c,const T *a, const T *b, size_t n,const dim3& grid,const dim3& block){
    size_t step = grid.x * block.x;
    add_kernel<T><<<grid,block>>>(c,a,b,n,step);
}


// Error checking macro
#define CUDA_CHECK(call)                                             \
{                                                                    \
    hipError_t err = call;                                          \
    if (err != hipSuccess)                                          \
{                                                                    \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                << " - " << hipGetErrorString(err) << "\n";         \
        exit(1);                                                     \
}                                                                    \
}

// Main program
int main()
{
    // 1.Prepare and initialize data (CPU)
    const size_t SIZE = 1<<20;
	size_t size_bytes = SIZE*sizeof(float);

	// Allocate memory for arrays A, B, and C on host
    // 1.1 by malloc
	// double *A = (double*)malloc(bytes);
	// double *B = (double*)malloc(bytes);
	// double *C = (double*)malloc(bytes);

    // 1.2 by vector
    std::vector<float> h_a(SIZE,1);
    std::vector<float> h_b(SIZE,2);
    std::vector<float> h_c(SIZE,0);

    // 2.Transfer data to GPU
	// 2.1 Allocate memory for arrays d_a, d_b, and d_c on device
	float *d_a, *d_b, *d_c;
	CUDA_CHECK(hipMalloc(&d_a, size_bytes));
	CUDA_CHECK(hipMalloc(&d_b, size_bytes));
	CUDA_CHECK(hipMalloc(&d_c, size_bytes));

	// 2.2 Copy data from host to device arrays 
	CUDA_CHECK(hipMemcpy(d_a, h_a.data(), size_bytes, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_b, h_b.data(), size_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_c, h_c.data(), size_bytes, hipMemcpyHostToDevice));

	// Set execution configuration parameters
	//		grid_dim: number of CUDA threads per grid block
	//		block_dim: number of blocks in grid
	dim3 block_dim(256);
	dim3 grid_dim(256);


    // 3.GPU reads from global memory, performs computation, and writes back (invoke computation function)
	// call the cuda add kernel
	vector_add( d_c,d_a, d_b,SIZE,grid_dim,block_dim);

    // 4.Transfer data back from GPU to CPU
	// Copy data from device 
	CUDA_CHECK(hipMemcpy(h_c.data(), d_c, size_bytes, hipMemcpyDeviceToHost));

    if (d_a)
        CUDA_CHECK(hipFree(d_a));
    if (d_b)
        CUDA_CHECK(hipFree(d_b));
    if (d_c)
        CUDA_CHECK(hipFree(d_c));
	// 5.Verify results
    std::cout << "执行完毕" << '\n'; 
    std::cout <<"c[SIZE-1]:"<< h_c[SIZE-1] << '\n'; 
	return 0;
}
